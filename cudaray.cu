#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include "cuda_kernels.h"
#include "cuda_vec3.h"
#include "cuda_ray.h"
#include "cuda_sphere.h"
#include "cuda_camera.h"
#include "cuda_material.h"
#include "stb_image.h"
#include "cuda_texobj.h"
#include <vector>
#include <random>
#include <iostream>              // <<< add this right after the other #includes


extern __constant__ hipTextureObject_t dev_textures[5];

// Define constant memory as a byte buffer to avoid constructor issues
__constant__ unsigned char const_spheres_buffer[64 * sizeof(Sphere)];

// FINAL RUN PARAMS
// #define WIDTH 1920
// #define HEIGHT 1080
// #define SAMPLES_PER_PIXEL 300
// #define MAX_DEPTH 30
// #define NUM_SPHERES 25

#define WIDTH 1280
#define HEIGHT 720
#define SAMPLES_PER_PIXEL 30
#define MAX_DEPTH 10
#define NUM_SPHERES 3
#define MAX_SPHERES 64 

__host__ void save_image(const char* filename, unsigned char* image) {
    FILE* fp = fopen(filename, "w");
    fprintf(fp, "P3\n%d %d\n255\n", WIDTH, HEIGHT);
    for (int i = 0; i < WIDTH * HEIGHT * 3; i += 3) {
        fprintf(fp, "%d %d %d\n", image[i], image[i+1], image[i+2]);
    }
    fclose(fp);
}

void build_scene(std::vector<Sphere>& H)
{
    std::mt19937 gen{42};
    std::uniform_real_distribution<float> uni(0.0f, 1.0f);

    // --- ground ---
    H.push_back({vec3(0, -1000, 0), 1000.0f,           // radius  -> float
               CHECKER, vec3(1), 0.0f, 1.0f, -1});

    // --- five large spheres for texture testing ---
    vec3 textured_positions[5] = {
        vec3(-6.0f, 1.0f, 6.0f),
        vec3(-3.0f, 1.0f, 6.0f),
        vec3( 0.0f, 1.0f, 6.0f),
        vec3( 3.0f, 1.0f, 6.0f),
        vec3( 6.0f, 1.0f, 6.0f)
    };
    for (int i = 0; i < 5; ++i) {
    H.push_back({textured_positions[i], 1.0f, TEXTURED, vec3(1.0f), 0.0f, 1.0f, i}); // bind i-th texture
    }


    // --- two fixed known spheres (for control testing) ---
    H.push_back({vec3(-4, 1.0f, 0), 1.0f, DIELECTRIC,
               vec3(1), 0.0f, 1.5f,-1});                 // fuzz -> 0.0f

    H.push_back({vec3( 4, 1.0f, 0), 1.0f, DIELECTRIC,
                vec3(1), 0.0f, 1.5f, -1});



    // --- small random balls grid ---
    for (int a = -NUM_SPHERES; a <= NUM_SPHERES; ++a)
    {
        for (int b = -NUM_SPHERES; b <= NUM_SPHERES; ++b) {
            vec3 center(a + 0.9f * uni(gen), 0.2f, b + 0.9f * uni(gen));

            // Avoid overlapping the main spheres (control + textured)
            vec3 exclusion_centers[] = {
                vec3(-4,1,0), vec3(4,1,0),
                vec3(-6,1,6), vec3(-3,1,6), vec3(0,1,6), vec3(3,1,6), vec3(6,1,6)
            };

            bool skip = false;
            for (const vec3& p : exclusion_centers)
                if ((center - p).length() < 1.2f) { skip = true; break; }
            if (skip) continue;

            float choose = uni(gen);
            if (choose < 0.6f) {
                vec3 col = vec3(uni(gen), uni(gen), uni(gen));
                col = col * col;
                H.push_back({center, 0.2f, LAMBERTIAN, col,
                    0.0f, 1.0f,-1}); 
            } else if (choose < 0.85f) {
                vec3 col = 0.5f * vec3(1 + uni(gen), 1 + uni(gen), 1 + uni(gen));
                float fuzz = 0.01f + 0.08f * uni(gen);
                H.push_back({center, 0.2f, METAL, col, fuzz, 1.0f,-1});
            } else {
                H.push_back({center, 0.2f, DIELECTRIC,
                    vec3(1), 0.0f, 1.5f,-1});
            }
        }
    }
}

// TIMING TEMPLATE
template<typename KernelPtr>
float time_kernel_3D(KernelPtr k,
                     unsigned char* d_img,
                     Camera* d_cam,
                     Sphere* d_spheres,
                     int n, int maxDepth,
                     dim3 grid, dim3 block)
{
    hipEvent_t t0, t1;  hipEventCreate(&t0);  hipEventCreate(&t1);
    hipEventRecord(t0);
    k<<<grid,block>>>(d_img, d_cam, d_spheres, n, maxDepth);
    hipEventRecord(t1);
    hipEventSynchronize(t1);
    float ms=0; hipEventElapsedTime(&ms,t0,t1);
    return ms;
}

// overload for constant-memory kernel (no Sphere* param)
template<typename KernelPtr>
float time_kernel_const(KernelPtr k,
                        unsigned char* d_img,
                        Camera* d_cam,
                        int n, int maxDepth,
                        dim3 grid, dim3 block)
{
    hipEvent_t t0, t1;  hipEventCreate(&t0);  hipEventCreate(&t1);
    hipEventRecord(t0);
    k<<<grid,block>>>(d_img, d_cam, n, maxDepth);
    hipEventRecord(t1);
    hipEventSynchronize(t1);
    float ms=0; hipEventElapsedTime(&ms,t0,t1);
    return ms;
}


int main() {
    size_t img_size = WIDTH * HEIGHT * 3;
    unsigned char* h_img = (unsigned char*)malloc(img_size);
    unsigned char* d_img;
    // -----------TEXTURE MEMORY SETUP ---------//
    CudaTexture h_textures[5];
    unsigned char* h_tex_data[5];
    int tex_width[5], tex_height[5], tex_channels[5];

    const char* tex_files[5] = {
        "./test_textures/beach_probe.jpg",
        "./test_textures/building_probe.jpg",
        "./test_textures/campus_probe.jpg",
        "./test_textures/kitchen_probe.jpg",
        "./test_textures/tex.jpg"
    };

    for (int i = 0; i < 5; ++i) {
        h_tex_data[i] = stbi_load(tex_files[i], &tex_width[i], &tex_height[i], &tex_channels[i], 4);
        if (!h_tex_data[i]) {
            printf("Failed to load texture %s\n", tex_files[i]);
            exit(1);
        }

        hipChannelFormatDesc desc = hipCreateChannelDesc<uchar4>();
        hipMallocArray(&h_textures[i].array, &desc, tex_width[i], tex_height[i]);

        hipMemcpy2DToArray(h_textures[i].array, 0, 0, h_tex_data[i],
                            tex_width[i] * 4, tex_width[i] * 4, tex_height[i],
                            hipMemcpyHostToDevice);

        hipResourceDesc resDesc = {};
        resDesc.resType = hipResourceTypeArray;
        resDesc.res.array.array = h_textures[i].array;

        hipTextureDesc texDesc = {};
        texDesc.addressMode[0] = hipAddressModeWrap;
        texDesc.addressMode[1] = hipAddressModeWrap;
        texDesc.filterMode = hipFilterModeLinear;
        texDesc.readMode = hipReadModeNormalizedFloat;
        texDesc.normalizedCoords = 1;

        hipCreateTextureObject(&h_textures[i].texObj, &resDesc, &texDesc, nullptr);
    }
    hipTextureObject_t h_textureObjs[5];
    for (int i = 0; i < 5; ++i) {
        h_textureObjs[i] = h_textures[i].texObj;
    }

    hipMemcpyToSymbol(HIP_SYMBOL(dev_textures), h_textureObjs, sizeof(h_textureObjs));

    // ----------SCENE SETUP--------------- //
    vec3 lookFrom = vec3(20.0f, 5.0f, 20.0f);  // farther & slightly higher
    vec3 lookAt   = vec3(0.0f, 0.5f, 0.0f);   // aim just above ground
    vec3 up = {0.0f, 1.0f, 0.0f};

    float distToFocus = (lookFrom - lookAt).length();
    float aperture = 0.03f;
    float aspect_ratio = float(WIDTH) / HEIGHT;

    Camera h_cam(lookFrom, lookAt, up, 20.0f, aspect_ratio, aperture, distToFocus);
    Camera* d_cam;
    hipMalloc(&d_cam, sizeof(Camera));
    hipMemcpy(d_cam, &h_cam, sizeof(Camera), hipMemcpyHostToDevice);

    hipMalloc(&d_img, img_size);

    std::vector<Sphere> host_spheres;
    build_scene(host_spheres);
    
    // Debug: print sphere count and size
    std::cout << "Number of spheres: " << host_spheres.size() << std::endl;
    std::cout << "Size of Sphere: " << sizeof(Sphere) << " bytes" << std::endl;
    std::cout << "Total size needed: " << host_spheres.size() * sizeof(Sphere) << " bytes" << std::endl;
    std::cout << "Constant memory buffer size: " << sizeof(const_spheres_buffer) << " bytes" << std::endl;
    
    std::cout << "Total spheres in scene: " << host_spheres.size() << std::endl;
    if (host_spheres.size() > MAX_SPHERES) {
        std::cerr << "ERROR: Scene has " << host_spheres.size() 
                  << " spheres, but constant memory can only hold " << MAX_SPHERES << std::endl;
    }


    
    
    Sphere* d_spheres;
    hipMalloc(&d_spheres, host_spheres.size()*sizeof(Sphere));
    hipMemcpy(d_spheres, host_spheres.data(),
            host_spheres.size()*sizeof(Sphere), hipMemcpyHostToDevice);

    // ------------ copy the same scene to constant memory once ------------
    hipMemcpyToSymbol(HIP_SYMBOL(const_spheres_buffer),
        host_spheres.data(),
        host_spheres.size() * sizeof(Sphere));

    // TODO: Launch rayKernel here
    hipDeviceSetLimit(hipLimitStackSize, 32768); // or 32768 for deep recursion

    dim3 block2D(16,16);                                    // for original 2-D kernel
    dim3 grid2D((WIDTH+block2D.x-1)/block2D.x,
                (HEIGHT+block2D.y-1)/block2D.y);

    dim3 block1D(256);                                      // for shared/constant
    dim3 grid1D((WIDTH*HEIGHT + block1D.x - 1)/block1D.x);


    dim3 block(16, 16);
    dim3 grid((WIDTH + block.x - 1) / block.x, (HEIGHT + block.y - 1) / block.y);
    rayKernel<<<grid, block>>>(d_img, d_cam, d_spheres,
        static_cast<int>(host_spheres.size()),
        MAX_DEPTH);      // add arg
    hipDeviceSynchronize();


    float t_global = time_kernel_3D(rayKernel,           // your original
        d_img, d_cam, d_spheres,
        host_spheres.size(), MAX_DEPTH,
        grid2D, block2D);

    hipMemcpy(h_img, d_img, img_size, hipMemcpyDeviceToHost);
    save_image("out_global.ppm", h_img);
    std::cout << "Global   " << t_global << "  ms  (out_global.ppm)\n";

    // -------------------------------------------------------------------
    float t_const  = time_kernel_const(rayKernel_constant,
            d_img, d_cam,
            host_spheres.size(), MAX_DEPTH,
            grid1D, block1D);

    hipMemcpy(h_img, d_img, img_size, hipMemcpyDeviceToHost);
    save_image("out_const.ppm", h_img);
    std::cout << "Const    " << t_const  << "  ms  (out_const.ppm)\n";

    // summary
    std::cout << "Speed-up vs Global  →  Const: "
    << t_global/t_const  << "×\n";


    hipMemcpy(h_img, d_img, img_size, hipMemcpyDeviceToHost);

    hipFree(d_img);
    hipFree(d_cam);
    free(h_img);
    return 0;
}
